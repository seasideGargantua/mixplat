#include "hip/hip_runtime.h"
#include "bindings.h"
#include "helpers.cuh"
#include "auxiliary.h"
#include "types.cuh"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <iostream>
#include <hip/hip_fp16.h>

namespace mixplat {

namespace cg = cooperative_groups;

/****************************************************************************
 * Rasterization of Gaussians utils
 ****************************************************************************/

// kernel to map each intersection from tile ID and depth to a gaussian
// writes output to isect_ids and gaussian_ids
__global__ void map_gaussian_to_intersects(
    const int num_points,
    const float2* __restrict__ xys,
    const float* __restrict__ depths,
    const int* __restrict__ radii,
    const int32_t* __restrict__ cum_tiles_hit,
    const dim3 tile_bounds,
    const unsigned block_width,
    int64_t* __restrict__ isect_ids,
    int32_t* __restrict__ gaussian_ids
) {
    unsigned idx = cg::this_grid().thread_rank();
    if (idx >= num_points)
        return;
    if (radii[idx] <= 0)
        return;
    // get the tile bbox for gaussian
    uint2 tile_min, tile_max;
    float2 center = xys[idx];
    get_tile_bbox(center, radii[idx], tile_bounds, tile_min, tile_max, block_width);
    // printf("point %d, %d radius, min %d %d, max %d %d\n", idx, radii[idx],
    // tile_min.x, tile_min.y, tile_max.x, tile_max.y);

    // update the intersection info for all tiles this gaussian hits
    int32_t cur_idx = (idx == 0) ? 0 : cum_tiles_hit[idx - 1];
    // printf("point %d starting at %d\n", idx, cur_idx);
    int64_t depth_id = (int64_t) * (int32_t *)&(depths[idx]);
    for (int i = tile_min.y; i < tile_max.y; ++i) {
        for (int j = tile_min.x; j < tile_max.x; ++j) {
            // isect_id is tile ID and depth as int32
            int64_t tile_id = i * tile_bounds.x + j; // tile within image
            isect_ids[cur_idx] = (tile_id << 32) | depth_id; // tile | depth id
            gaussian_ids[cur_idx] = idx;                     // 3D gaussian id
            ++cur_idx; // handles gaussians that hit more than one tile
        }
    }
    // printf("point %d ending at %d\n", idx, cur_idx);
}

std::tuple<torch::Tensor, torch::Tensor> map_gaussian_to_intersects_tensor(
    const int num_points,
    const int num_intersects,
    const torch::Tensor &xys,
    const torch::Tensor &depths,
    const torch::Tensor &radii,
    const torch::Tensor &cum_tiles_hit,
    const std::tuple<int, int, int> tile_bounds,
    const unsigned block_width
) {
    DEVICE_GUARD(xys);
    CHECK_INPUT(xys);
    CHECK_INPUT(depths);
    CHECK_INPUT(radii);
    CHECK_INPUT(cum_tiles_hit);

    dim3 tile_bounds_dim3;
    tile_bounds_dim3.x = std::get<0>(tile_bounds);
    tile_bounds_dim3.y = std::get<1>(tile_bounds);
    tile_bounds_dim3.z = std::get<2>(tile_bounds);

    torch::Tensor gaussian_ids_unsorted =
        torch::zeros({num_intersects}, xys.options().dtype(torch::kInt32));
    torch::Tensor isect_ids_unsorted =
        torch::zeros({num_intersects}, xys.options().dtype(torch::kInt64));

    map_gaussian_to_intersects<<<
        (num_points + N_THREADS - 1) / N_THREADS,
        N_THREADS>>>(
        num_points,
        (float2 *)xys.contiguous().data_ptr<float>(),
        depths.contiguous().data_ptr<float>(),
        radii.contiguous().data_ptr<int32_t>(),
        cum_tiles_hit.contiguous().data_ptr<int32_t>(),
        tile_bounds_dim3,
        block_width,
        // Outputs.
        isect_ids_unsorted.contiguous().data_ptr<int64_t>(),
        gaussian_ids_unsorted.contiguous().data_ptr<int32_t>()
    );

    return std::make_tuple(isect_ids_unsorted, gaussian_ids_unsorted);
}

// kernel to map sorted intersection IDs to tile bins
// expect that intersection IDs are sorted by increasing tile ID
// i.e. intersections of a tile are in contiguous chunks
__global__ void get_tile_bin_edges(
    const int num_intersects, const int64_t* __restrict__ isect_ids_sorted, int2* __restrict__ tile_bins
) {
    unsigned idx = cg::this_grid().thread_rank();
    if (idx >= num_intersects)
        return;
    // save the indices where the tile_id changes
    int32_t cur_tile_idx = (int32_t)(isect_ids_sorted[idx] >> 32);
    if (idx == 0 || idx == num_intersects - 1) {
        if (idx == 0)
            tile_bins[cur_tile_idx].x = 0;
        if (idx == num_intersects - 1)
            tile_bins[cur_tile_idx].y = num_intersects;
    }
    if (idx == 0)
        return;
    int32_t prev_tile_idx = (int32_t)(isect_ids_sorted[idx - 1] >> 32);
    if (prev_tile_idx != cur_tile_idx) {
        tile_bins[prev_tile_idx].y = idx;
        tile_bins[cur_tile_idx].x = idx;
        return;
    }
}

torch::Tensor get_tile_bin_edges_tensor(
    int num_intersects, const torch::Tensor &isect_ids_sorted, 
    const std::tuple<int, int, int> tile_bounds
) {
    DEVICE_GUARD(isect_ids_sorted);
    CHECK_INPUT(isect_ids_sorted);
    int num_tiles = std::get<0>(tile_bounds) * std::get<1>(tile_bounds);
    torch::Tensor tile_bins = torch::zeros(
        {num_tiles, 2}, isect_ids_sorted.options().dtype(torch::kInt32)
    );
    get_tile_bin_edges<<<
        (num_intersects + N_THREADS - 1) / N_THREADS,
        N_THREADS>>>(
        num_intersects,
        isect_ids_sorted.contiguous().data_ptr<int64_t>(),
        (int2 *)tile_bins.contiguous().data_ptr<int>()
    );
    return tile_bins;
}

}